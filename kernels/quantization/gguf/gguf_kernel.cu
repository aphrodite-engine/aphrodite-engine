#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

#include <torch/all.h>
#include <c10/cuda/CUDAGuard.h>

#include "cuda_compat.h"
#include "dispatch_utils.h"

#include "ggml-common.h"
#include "vecdotq.cuh"
#include "dequantize.cuh"
#include "mmvq.cuh"
#include "mmq.cuh"
#include "moe.cuh"

// Dynamic MMQ optimization
struct mmq_args {
    const void * x;
    const void * y;
    void * dst;
    int64_t nrows_x;
    int64_t ncols_x;
    int64_t nrows_y;
    int64_t ncols_y;
    int64_t nrows_dst;
};

template<int type>
static size_t get_mmq_nbytes_shared(int mmq_x, int mmq_y, int cc) {
    if (type == 12) { // Q4_K
        // Based on actual allocate_tiles_q4_K<mmq_y> function:
        // tile_x_ql: mmq_y * (WARP_SIZE_GGUF)       + mmq_y = mmq_y * (32 + 1) = mmq_y * 33
        // tile_x_dm: mmq_y * (WARP_SIZE_GGUF/QI4_K) + mmq_y/QI4_K = mmq_y * (32/8 + 1/8) = mmq_y * 4.125
        // tile_x_sc: mmq_y * (WARP_SIZE_GGUF/8)     + mmq_y/8 = mmq_y * (32/8 + 1/8) = mmq_y * 4.125
        // 
        // From mul_mat_q template:
        // tile_y_qs: mmq_x * WARP_SIZE_GGUF = mmq_x * 32
        // tile_y_ds: mmq_x * WARP_SIZE_GGUF/QI8_1 = mmq_x * (32/8) = mmq_x * 4

        const int tile_x_ql_size = mmq_y * (32 + 1) * sizeof(int);
        const int tile_x_dm_size = mmq_y * (32/8 + 1) * sizeof(half2);  // QI4_K = 8
        const int tile_x_sc_size = mmq_y * (32/8 + 1) * sizeof(int);    // /8 for Q4_K scales
        const int tile_y_qs_size = mmq_x * 32 * sizeof(int);
        const int tile_y_ds_size = mmq_x * (32/8) * sizeof(half2);      // QI8_1 = 8
        
        return tile_x_ql_size + tile_x_dm_size + tile_x_sc_size + tile_y_qs_size + tile_y_ds_size;
    }
    // Fallback for other types
    return mmq_x * mmq_y * sizeof(float) * 4;
}

// Dynamic kernel launcher template
template<typename scalar_t, int qtype, int mmq_x>
static void launch_mmq_kernel_optimized(const mmq_args& args, hipStream_t stream) {
    const int mmq_y = MMQ_Y_Q4_K;  // consistent MMQ_Y 
    const int nwarps = NWARPS_Q4_K;  // consistent NWARPS

    const int block_num_x = (args.nrows_x + mmq_y - 1) / mmq_y;
    const int block_num_y = (args.ncols_y + mmq_x - 1) / mmq_x;
    const dim3 block_nums(block_num_x, block_num_y, 1);
    const dim3 block_dims(32, nwarps, 1);  // WARP_SIZE_GGUF = 32

    if (qtype == 12) { // Q4_K only
        if (args.nrows_x % mmq_y == 0) {
            mul_mat_q4_K_dynamic<scalar_t, false, mmq_x><<<block_nums, block_dims, 0, stream>>>(
                args.x, args.y, (scalar_t*)args.dst, args.ncols_x, args.nrows_x, 
                args.ncols_y, args.nrows_y, args.nrows_dst);
        } else {
            mul_mat_q4_K_dynamic<scalar_t, true, mmq_x><<<block_nums, block_dims, 0, stream>>>(
                args.x, args.y, (scalar_t*)args.dst, args.ncols_x, args.nrows_x,
                args.ncols_y, args.nrows_y, args.nrows_dst);
        }
    }
    // Note: Q5_K and Q6_K not supported in this launcher - use original functions
}

// Optimized Q4_K kernel dispatch
template<typename scalar_t>
static void ggml_mul_mat_q4_K_q8_1_cuda_optimized(
    const void * vx, const void * vy, scalar_t * dst, 
    const int ncols_x, const int nrows_x, const int ncols_y, 
    const int nrows_y, const int nrows_dst, hipStream_t stream) {

    mmq_args args = {vx, vy, dst, nrows_x, ncols_x, nrows_y, ncols_y, nrows_dst};

    // Get device compute capability
    int device;
    hipGetDevice(&device);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);
    int cc = prop.major * 100 + prop.minor * 10;

    // CONSERVATIVE optimization
    // otherwise, we get NaN outputs
    // TODO: figure out what's happening here
    const int mmq_y = MMQ_Y_Q4_K;  // consistent MMQ_Y

    int mmq_x_best = MMQ_X_Q4_K;  // consistent MMQ_X

    // Only try modest improvements on modern GPUs for large batches
    const bool large_batch = ncols_y >= 512;
    if (cc >= 800 && large_batch) {  // only Ampere+ and large batches
        // test larger tile sizes for better performance
        for (int mmq_x : {8, 16, 24, 32, 40, 48, 56, 64}) {
            // Check shared memory constraints - now with accurate calculation
            const size_t shmem_needed = get_mmq_nbytes_shared<12>(mmq_x, mmq_y, cc);
            const size_t shmem_limit = (size_t)(prop.sharedMemPerBlock * 0.8); // Use 80% with accurate calculation
            if (shmem_needed <= shmem_limit) {
                const int ntiles_x = (ncols_y + mmq_x - 1) / mmq_x;
                const int ntiles_x_orig = (ncols_y + MMQ_X_Q4_K - 1) / MMQ_X_Q4_K;

                // Only use if it actually reduces the number of tiles
                if (ntiles_x < ntiles_x_orig) {
                    mmq_x_best = mmq_x;
                }
            }
        }
    }
    
    switch (mmq_x_best) {
        case 8:  launch_mmq_kernel_optimized<scalar_t, 12, 8>(args, stream); break;
        case 16: launch_mmq_kernel_optimized<scalar_t, 12, 16>(args, stream); break;
        case 24: launch_mmq_kernel_optimized<scalar_t, 12, 24>(args, stream); break;
        case 32: launch_mmq_kernel_optimized<scalar_t, 12, 32>(args, stream); break;
        case 40: launch_mmq_kernel_optimized<scalar_t, 12, 40>(args, stream); break;
        case 48: launch_mmq_kernel_optimized<scalar_t, 12, 48>(args, stream); break;
        case 56: launch_mmq_kernel_optimized<scalar_t, 12, 56>(args, stream); break;
        case 64: launch_mmq_kernel_optimized<scalar_t, 12, 64>(args, stream); break;
        default:
            // fallback to original implementation for safety
            ggml_mul_mat_q4_K_q8_1_cuda(vx, vy, dst, ncols_x, nrows_x, ncols_y, nrows_y, nrows_dst, stream);
            break;
    }
}

// Optimized Q5_K kernel dispatch
template<typename scalar_t>
static void ggml_mul_mat_q5_K_q8_1_cuda_optimized(
    const void * vx, const void * vy, scalar_t * dst, 
    const int ncols_x, const int nrows_x, const int ncols_y, 
    const int nrows_y, const int nrows_dst, hipStream_t stream) {

    // For now, disable optimization for Q5_K to ensure stability
    // TODO: Enable conservative optimization after Q4_K is fully validated
    ggml_mul_mat_q5_K_q8_1_cuda(vx, vy, dst, ncols_x, nrows_x, ncols_y, nrows_y, nrows_dst, stream);
}

// Optimized Q6_K kernel dispatch
template<typename scalar_t>
static void ggml_mul_mat_q6_K_q8_1_cuda_optimized(
    const void * vx, const void * vy, scalar_t * dst, 
    const int ncols_x, const int nrows_x, const int ncols_y, 
    const int nrows_y, const int nrows_dst, hipStream_t stream) {

    // For now, disable optimization for Q6_K to ensure stability
    // TODO: Enable conservative optimization after Q4_K is fully validated
    ggml_mul_mat_q6_K_q8_1_cuda(vx, vy, dst, ncols_x, nrows_x, ncols_y, nrows_y, nrows_dst, stream);
}

// Q8 gemv
template <typename scalar_t>
static __global__ void quantize_q8_1(const scalar_t* __restrict__ x,
                                     void* __restrict__ vy, const int kx,
                                     const int kx_padded) {
  const auto ix = blockDim.x * blockIdx.x + threadIdx.x;
  if (ix >= kx_padded) {
    return;
  }
  const auto iy = blockDim.y * blockIdx.y + threadIdx.y;
  const int i_padded = iy * kx_padded + ix;

  block_q8_1* y = (block_q8_1*)vy;

  const int ib = i_padded / QK8_1;   // block index
  const int iqs = i_padded % QK8_1;  // quant index

  const float xi = ix < kx ? static_cast<float>(x[iy * kx + ix]) : 0.0f;
  float amax = fabsf(xi);
  float sum = xi;

#pragma unroll
  for (int mask = 16; mask > 0; mask >>= 1) {
    amax = fmaxf(amax, APHRODITE_SHFL_XOR_SYNC_WIDTH(amax, mask, 32));
    sum += APHRODITE_SHFL_XOR_SYNC_WIDTH(sum, mask, 32);
  }

  const float d = amax / 127;
  const int8_t q = amax == 0.0f ? 0 : roundf(xi / d);

  y[ib].qs[iqs] = q;

  if (iqs > 0) {
    return;
  }

  y[ib].ds.x = __float2half(d);
  y[ib].ds.y = __float2half(sum);
}

template <typename scalar_t>
static void quantize_row_q8_1_cuda(const scalar_t* x, void* vy, const int kx,
                                   const int ky, hipStream_t stream) {
  const int64_t kx_padded = (kx + 512 - 1) / 512 * 512;
  const int block_num_x =
      (kx_padded + CUDA_QUANTIZE_BLOCK_SIZE - 1) / CUDA_QUANTIZE_BLOCK_SIZE;
  constexpr int MAX_BLOCK_SIZE = 65535;
  for (int off = 0; off < ky; off += MAX_BLOCK_SIZE) {
    const int num_blocks_y = std::min(ky, off + MAX_BLOCK_SIZE) - off;
    const dim3 num_blocks(block_num_x, num_blocks_y, 1);
    const dim3 block_size(CUDA_DEQUANTIZE_BLOCK_SIZE, 1, 1);
    quantize_q8_1<<<num_blocks, block_size, 0, stream>>>(
        &x[off * kx], (int32_t*)vy + off * (kx_padded / 32 * 9), kx, kx_padded);
  }
}

torch::Tensor ggml_dequantize(torch::Tensor W,  // quant weight
                              int64_t type, int64_t m, int64_t n,
                              std::optional<at::ScalarType> const& dtype) {
  const at::cuda::OptionalCUDAGuard device_guard(device_of(W));
  auto dtype_ = dtype.value_or(torch::kFloat16);
  auto options = torch::TensorOptions().dtype(dtype_).device(W.device());
  at::Tensor DW = torch::empty({m, n}, options);
  hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();

  APHRODITE_DISPATCH_FLOATING_TYPES(DW.scalar_type(), "ggml_dequantize", [&] {
    auto to_cuda = ggml_get_to_cuda<scalar_t>(type);
    to_cuda((void*)W.data_ptr(), (scalar_t*)DW.data_ptr(), m * n, stream);
  });

  return DW;
}

torch::Tensor ggml_mul_mat_vec_a8(torch::Tensor W,  // quant weight
                                  torch::Tensor X,  // input
                                  int64_t type, int64_t row) {
  int col = X.sizes()[1];
  const int padded = (col + 512 - 1) / 512 * 512;
  const at::cuda::OptionalCUDAGuard device_guard(device_of(X));
  auto options = torch::TensorOptions().dtype(X.dtype()).device(W.device());
  at::Tensor Y = torch::empty({1, row}, options);
  hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();
  options = torch::TensorOptions().dtype(torch::kInt32).device(W.device());
  at::Tensor quant_X = torch::empty({1, padded / 32 * 9}, options);
  APHRODITE_DISPATCH_FLOATING_TYPES(
      X.scalar_type(), "ggml_mul_mat_vec_a8", [&] {
        quantize_row_q8_1_cuda<scalar_t>(
            (scalar_t*)X.data_ptr(), (void*)quant_X.data_ptr(), col, 1, stream);
        switch (type) {
          case 2:
            mul_mat_vec_q4_0_q8_1_cuda<scalar_t>(
                (void*)W.data_ptr(), (void*)quant_X.data_ptr(),
                (scalar_t*)Y.data_ptr(), col, row, stream);
            break;
          case 3:
            mul_mat_vec_q4_1_q8_1_cuda<scalar_t>(
                (void*)W.data_ptr(), (void*)quant_X.data_ptr(),
                (scalar_t*)Y.data_ptr(), col, row, stream);
            break;
          case 6:
            mul_mat_vec_q5_0_q8_1_cuda<scalar_t>(
                (void*)W.data_ptr(), (void*)quant_X.data_ptr(),
                (scalar_t*)Y.data_ptr(), col, row, stream);
            break;
          case 7:
            mul_mat_vec_q5_1_q8_1_cuda<scalar_t>(
                (void*)W.data_ptr(), (void*)quant_X.data_ptr(),
                (scalar_t*)Y.data_ptr(), col, row, stream);
            break;
          case 8:
            mul_mat_vec_q8_0_q8_1_cuda<scalar_t>(
                (void*)W.data_ptr(), (void*)quant_X.data_ptr(),
                (scalar_t*)Y.data_ptr(), col, row, stream);
            break;
          case 10:
            mul_mat_vec_q2_K_q8_1_cuda<scalar_t>(
                (void*)W.data_ptr(), (void*)quant_X.data_ptr(),
                (scalar_t*)Y.data_ptr(), col, row, stream);
            break;
          case 11:
            mul_mat_vec_q3_K_q8_1_cuda<scalar_t>(
                (void*)W.data_ptr(), (void*)quant_X.data_ptr(),
                (scalar_t*)Y.data_ptr(), col, row, stream);
            break;
          case 12:
            mul_mat_vec_q4_K_q8_1_cuda<scalar_t>(
                (void*)W.data_ptr(), (void*)quant_X.data_ptr(),
                (scalar_t*)Y.data_ptr(), col, row, stream);
            break;
          case 13:
            mul_mat_vec_q5_K_q8_1_cuda<scalar_t>(
                (void*)W.data_ptr(), (void*)quant_X.data_ptr(),
                (scalar_t*)Y.data_ptr(), col, row, stream);
            break;
          case 14:
            mul_mat_vec_q6_K_q8_1_cuda<scalar_t>(
                (void*)W.data_ptr(), (void*)quant_X.data_ptr(),
                (scalar_t*)Y.data_ptr(), col, row, stream);
            break;
          case 16:
            mul_mat_vec_iq2_xxs_q8_1_cuda<scalar_t>(
                (void*)W.data_ptr(), (void*)quant_X.data_ptr(),
                (scalar_t*)Y.data_ptr(), col, row, stream);
            break;
          case 17:
            mul_mat_vec_iq2_xs_q8_1_cuda<scalar_t>(
                (void*)W.data_ptr(), (void*)quant_X.data_ptr(),
                (scalar_t*)Y.data_ptr(), col, row, stream);
            break;
          case 18:
            mul_mat_vec_iq3_xxs_q8_1_cuda<scalar_t>(
                (void*)W.data_ptr(), (void*)quant_X.data_ptr(),
                (scalar_t*)Y.data_ptr(), col, row, stream);
            break;
          case 19:
            mul_mat_vec_iq1_s_q8_1_cuda<scalar_t>(
                (void*)W.data_ptr(), (void*)quant_X.data_ptr(),
                (scalar_t*)Y.data_ptr(), col, row, stream);
            break;
          case 20:
            mul_mat_vec_iq4_nl_q8_1_cuda<scalar_t>(
                (void*)W.data_ptr(), (void*)quant_X.data_ptr(),
                (scalar_t*)Y.data_ptr(), col, row, stream);
            break;
          case 21:
            mul_mat_vec_iq3_s_q8_1_cuda<scalar_t>(
                (void*)W.data_ptr(), (void*)quant_X.data_ptr(),
                (scalar_t*)Y.data_ptr(), col, row, stream);
            break;
          case 22:
            mul_mat_vec_iq2_s_q8_1_cuda<scalar_t>(
                (void*)W.data_ptr(), (void*)quant_X.data_ptr(),
                (scalar_t*)Y.data_ptr(), col, row, stream);
            break;
          case 23:
            mul_mat_vec_iq4_xs_q8_1_cuda<scalar_t>(
                (void*)W.data_ptr(), (void*)quant_X.data_ptr(),
                (scalar_t*)Y.data_ptr(), col, row, stream);
            break;
          case 29:
            mul_mat_vec_iq1_m_q8_1_cuda<scalar_t>(
                (void*)W.data_ptr(), (void*)quant_X.data_ptr(),
                (scalar_t*)Y.data_ptr(), col, row, stream);
            break;
        }
      });
  return Y;
}

torch::Tensor ggml_mul_mat_a8(torch::Tensor W,  // quant weight
                              torch::Tensor X,  // input
                              int64_t type, int64_t row) {
  int col = X.sizes()[1];
  int padded = (col + 512 - 1) / 512 * 512;
  int batch = X.sizes()[0];
  const at::cuda::OptionalCUDAGuard device_guard(device_of(X));
  auto options = torch::TensorOptions().dtype(X.dtype()).device(W.device());
  at::Tensor Y = torch::empty({batch, row}, options);
  hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();
  options = torch::TensorOptions().dtype(torch::kInt32).device(W.device());
  at::Tensor quant_X = torch::empty({batch, padded / 32 * 9}, options);
  APHRODITE_DISPATCH_FLOATING_TYPES(X.scalar_type(), "ggml_mul_mat_a8", [&] {
    quantize_row_q8_1_cuda((scalar_t*)X.data_ptr(), (void*)quant_X.data_ptr(),
                           col, batch, stream);

    switch (type) {
      case 2:
        ggml_mul_mat_q4_0_q8_1_cuda(
            (void*)W.data_ptr(), (void*)quant_X.data_ptr(),
            (scalar_t*)Y.data_ptr(), col, row, batch, padded, row, stream);
        break;
      case 3:
        ggml_mul_mat_q4_1_q8_1_cuda(
            (void*)W.data_ptr(), (void*)quant_X.data_ptr(),
            (scalar_t*)Y.data_ptr(), col, row, batch, padded, row, stream);
        break;
      case 6:
        ggml_mul_mat_q5_0_q8_1_cuda(
            (void*)W.data_ptr(), (void*)quant_X.data_ptr(),
            (scalar_t*)Y.data_ptr(), col, row, batch, padded, row, stream);
        break;
      case 7:
        ggml_mul_mat_q5_1_q8_1_cuda(
            (void*)W.data_ptr(), (void*)quant_X.data_ptr(),
            (scalar_t*)Y.data_ptr(), col, row, batch, padded, row, stream);
        break;
      case 8:
        ggml_mul_mat_q8_0_q8_1_cuda(
            (void*)W.data_ptr(), (void*)quant_X.data_ptr(),
            (scalar_t*)Y.data_ptr(), col, row, batch, padded, row, stream);
        break;
      case 10:
        ggml_mul_mat_q2_K_q8_1_cuda(
            (void*)W.data_ptr(), (void*)quant_X.data_ptr(),
            (scalar_t*)Y.data_ptr(), col, row, batch, padded, row, stream);
        break;
      case 11:
        ggml_mul_mat_q3_K_q8_1_cuda(
            (void*)W.data_ptr(), (void*)quant_X.data_ptr(),
            (scalar_t*)Y.data_ptr(), col, row, batch, padded, row, stream);
        break;
      case 12:
        ggml_mul_mat_q4_K_q8_1_cuda_optimized(
            (void*)W.data_ptr(), (void*)quant_X.data_ptr(),
            (scalar_t*)Y.data_ptr(), col, row, batch, padded, row, stream);
        break;
      case 13:
        ggml_mul_mat_q5_K_q8_1_cuda_optimized(
            (void*)W.data_ptr(), (void*)quant_X.data_ptr(),
            (scalar_t*)Y.data_ptr(), col, row, batch, padded, row, stream);
        break;
      case 14:
        ggml_mul_mat_q6_K_q8_1_cuda_optimized(
            (void*)W.data_ptr(), (void*)quant_X.data_ptr(),
            (scalar_t*)Y.data_ptr(), col, row, batch, padded, row, stream);
        break;
    }
  });
  return Y;
}

torch::Tensor ggml_moe_a8(torch::Tensor X,  // input
                          torch::Tensor W,  // expert weights
                          torch::Tensor sorted_token_ids,
                          torch::Tensor expert_ids,
                          torch::Tensor num_tokens_post_padded, int64_t type,
                          int64_t row, int64_t top_k, int64_t tokens) {
  int col = X.sizes()[1];
  int padded = (col + 512 - 1) / 512 * 512;
  const at::cuda::OptionalCUDAGuard device_guard(device_of(X));
  auto options = torch::TensorOptions().dtype(X.dtype()).device(W.device());
  at::Tensor Y = torch::empty({tokens * top_k, row}, options);
  hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();
  options = torch::TensorOptions().dtype(torch::kInt32).device(W.device());
  at::Tensor quant_X = torch::empty({tokens, padded / 32 * 9}, options);
  APHRODITE_DISPATCH_FLOATING_TYPES(X.scalar_type(), "ggml_moe_a8", [&] {
    quantize_row_q8_1_cuda((scalar_t*)X.data_ptr(), (void*)quant_X.data_ptr(),
                           col, tokens, stream);
    switch (type) {
      case 2:
        ggml_moe_q4_0_q8_1_cuda(
            (void*)quant_X.data_ptr(), (void*)W.data_ptr(),
            (scalar_t*)Y.data_ptr(), (int*)sorted_token_ids.data_ptr(),
            (int*)expert_ids.data_ptr(),
            (int*)num_tokens_post_padded.data_ptr(), W.stride(0), col, row,
            tokens, padded, row, top_k, sorted_token_ids.sizes()[0], stream);
        break;
      case 3:
        ggml_moe_q4_1_q8_1_cuda(
            (void*)quant_X.data_ptr(), (void*)W.data_ptr(),
            (scalar_t*)Y.data_ptr(), (int*)sorted_token_ids.data_ptr(),
            (int*)expert_ids.data_ptr(),
            (int*)num_tokens_post_padded.data_ptr(), W.stride(0), col, row,
            tokens, padded, row, top_k, sorted_token_ids.sizes()[0], stream);
        break;
      case 6:
        ggml_moe_q5_0_q8_1_cuda(
            (void*)quant_X.data_ptr(), (void*)W.data_ptr(),
            (scalar_t*)Y.data_ptr(), (int*)sorted_token_ids.data_ptr(),
            (int*)expert_ids.data_ptr(),
            (int*)num_tokens_post_padded.data_ptr(), W.stride(0), col, row,
            tokens, padded, row, top_k, sorted_token_ids.sizes()[0], stream);
        break;
      case 7:
        ggml_moe_q5_1_q8_1_cuda(
            (void*)quant_X.data_ptr(), (void*)W.data_ptr(),
            (scalar_t*)Y.data_ptr(), (int*)sorted_token_ids.data_ptr(),
            (int*)expert_ids.data_ptr(),
            (int*)num_tokens_post_padded.data_ptr(), W.stride(0), col, row,
            tokens, padded, row, top_k, sorted_token_ids.sizes()[0], stream);
        break;
      case 8:
        ggml_moe_q8_0_q8_1_cuda(
            (void*)quant_X.data_ptr(), (void*)W.data_ptr(),
            (scalar_t*)Y.data_ptr(), (int*)sorted_token_ids.data_ptr(),
            (int*)expert_ids.data_ptr(),
            (int*)num_tokens_post_padded.data_ptr(), W.stride(0), col, row,
            tokens, padded, row, top_k, sorted_token_ids.sizes()[0], stream);
        break;
      case 10:
        ggml_moe_q2_K_q8_1_cuda(
            (void*)quant_X.data_ptr(), (void*)W.data_ptr(),
            (scalar_t*)Y.data_ptr(), (int*)sorted_token_ids.data_ptr(),
            (int*)expert_ids.data_ptr(),
            (int*)num_tokens_post_padded.data_ptr(), W.stride(0), col, row,
            tokens, padded, row, top_k, sorted_token_ids.sizes()[0], stream);
        break;
      case 11:
        ggml_moe_q3_K_q8_1_cuda(
            (void*)quant_X.data_ptr(), (void*)W.data_ptr(),
            (scalar_t*)Y.data_ptr(), (int*)sorted_token_ids.data_ptr(),
            (int*)expert_ids.data_ptr(),
            (int*)num_tokens_post_padded.data_ptr(), W.stride(0), col, row,
            tokens, padded, row, top_k, sorted_token_ids.sizes()[0], stream);
        break;
      case 12:
        ggml_moe_q4_K_q8_1_cuda(
            (void*)quant_X.data_ptr(), (void*)W.data_ptr(),
            (scalar_t*)Y.data_ptr(), (int*)sorted_token_ids.data_ptr(),
            (int*)expert_ids.data_ptr(),
            (int*)num_tokens_post_padded.data_ptr(), W.stride(0), col, row,
            tokens, padded, row, top_k, sorted_token_ids.sizes()[0], stream);
        break;
      case 13:
        ggml_moe_q5_K_q8_1_cuda(
            (void*)quant_X.data_ptr(), (void*)W.data_ptr(),
            (scalar_t*)Y.data_ptr(), (int*)sorted_token_ids.data_ptr(),
            (int*)expert_ids.data_ptr(),
            (int*)num_tokens_post_padded.data_ptr(), W.stride(0), col, row,
            tokens, padded, row, top_k, sorted_token_ids.sizes()[0], stream);
        break;
      case 14:
        ggml_moe_q6_K_q8_1_cuda(
            (void*)quant_X.data_ptr(), (void*)W.data_ptr(),
            (scalar_t*)Y.data_ptr(), (int*)sorted_token_ids.data_ptr(),
            (int*)expert_ids.data_ptr(),
            (int*)num_tokens_post_padded.data_ptr(), W.stride(0), col, row,
            tokens, padded, row, top_k, sorted_token_ids.sizes()[0], stream);
        break;
    }
  });
  return Y;
}

int64_t ggml_moe_get_block_size(int64_t type) {
  switch (type) {
    case 2:
      return MOE_X_Q4_0;
    case 3:
      return MOE_X_Q4_1;
    case 6:
      return MOE_X_Q5_0;
    case 7:
      return MOE_X_Q5_1;
    case 8:
      return MOE_X_Q8_0;
    case 10:
      return MOE_X_Q2_K;
    case 11:
      return MOE_X_Q3_K;
    case 12:
      return MOE_X_Q4_K;
    case 13:
      return MOE_X_Q5_K;
    case 14:
      return MOE_X_Q6_K;
  }
  return 0;
}
